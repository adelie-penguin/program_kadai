/* vim: set tabstop=4 foldmethod=marker : */
/************************************************
 *
 * CUDAおためしプログラム
 *   ・CPU-GPU間のデータ転送
 *   ・シェアードメモリの利用
 *
 *-----------------------------------------------
 *
 *  コンパイルコマンド
 *   $nvcc -o cuda02 cuda02.cu
 *
 *  実行コマンド
 *   $./cuda02
 *
 ************************************************/

#include<stdio.h>
#include<hip/hip_runtime.h>

#define ThreadMax (10)
#define MallSize  ((ThreadMax) * sizeof(int))

/*==============================================*/
/* GPUで実行する関数                            */
/*----------------------------------------------*/
/* ※シェアードメモリをわざと経由                */
/*==============================================*/
__global__ void test_asemble(int *g_data, int num)
{
	int idx = threadIdx.x;      /* レジスタ変数(スレッドIDの取得)           */
	__shared__ int  s_data;     /* シェアード変数                           */

	s_data      = g_data[idx];  /* グローバルメモリからシェアードにコピー   */
	s_data     += idx * num;
	g_data[idx] = s_data;       /* グローバルメモリへの書き戻し             */

	return;
}

/************************************************/
/* main関数(CPUで実行)                          */
/************************************************/
int main(int argc, char *argv[])
{
	int i;
	int *c_array   = NULL;  /* CPU側の配列 */
	int *g_array   = NULL;  /* GPU側の配列 */
	int thread_max = 10;
	int grid_max   = 1;


	/*--[スレッド数とブロック数を決める]-------*/

	dim3 grid(grid_max);            /* スレッドブロックの個数を指定         */
	dim3 block(thread_max);         /* スレブロのサイズ(スレッド数)を指定   */


	/*--[CPU側のメモリ確保]--------------------*/

	c_array = (int*)malloc(thread_max * sizeof(int));   /* CPU上にメモリ確保 */

	for(int i = 0; i < thread_max; i++)           
	{
		c_array[i] = 0;     /* 配列初期化 */
	}

	/*--[CUDA側のメモリ確保]-------------------*/

	/* GPU上にメモリ確保        */
	hipMalloc((int**)&g_array, MallSize);

	/* CPUの配列をGPUにコピー   */
	hipMemcpy(g_array, c_array, MallSize, hipMemcpyHostToDevice);


	/*--[GPUのカーネル起動]--------------------*/

	test_asemble<<<grid, block>>>(g_array, 10);


	/*--[仕事結果の回収]-----------------------*/

	/* GPUの配列をCPUにコピー   */
	hipMemcpy(c_array, g_array, MallSize, hipMemcpyDeviceToHost);

	/* 表示 */
	for(i = 0; i < thread_max; i++)
	{
		fprintf(stdout, "c_array[%d]=%d\n", i, c_array[i]);
	}

	hipFree(g_array);
	free(c_array);
	hipDeviceReset(); /* デバイスリセット */
	return(0);
}
