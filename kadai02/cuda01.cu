/* vim: set tabstop=4 : */
/****************************************************************************
 * cuda のおためしプログラム
 *----------------------------------------------------------------------
 *
 * コンパイルコマンド
 * $make
 * 実行コマンド
 * $./cuda01 
 *
 ****************************************************************************/

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#define BLOCK_MAX   2	/* スレッドブロック数 */
#define BLOCK_SIZE  4	/* スレッド数 */

__global__ void function(void);

int main(int argc, char *argv[])
{
	function<<<BLOCK_MAX, BLOCK_SIZE>>>();
	hipDeviceReset();
	return(0);
}

__global__ void function(void)
{
	printf(">> block%d thread%d \n", blockIdx.x, threadIdx.x);
}
