/* vim: set tabstop=4 : */
/****************************************************************************
 * cuda のおためしプログラム
 *----------------------------------------------------------------------
 *
 * コンパイルコマンド
 * $make
 * 実行コマンド
 * $./cuda2 
 *
 ****************************************************************************/
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

/* CUDAが無効な状態でも実行したい場合はこのifdefに代替コードを記述 */
#ifdef __HIPCC__
#include<hip/hip_runtime.h>
#define penguin_print()	fprintf(stdout, ">> with cuda\n")
#else
#define penguin_print()	fprintf(stdout, ">> without cuda\n")
#endif
/*------------*/

#define BLOCK_MAX   2	/* スレッドブロック数 */
#define BLOCK_SIZE  4	/* スレッド数 */

__global__ void function(void);

int main(int argc, char *argv[])
{
	penguin_print();
	function<<<BLOCK_MAX, BLOCK_SIZE>>>();
	hipDeviceReset();
	return(0);
}

__global__ void function(void)
{
	printf(">> block%d thread%d \n", blockIdx.x, threadIdx.x);
}
